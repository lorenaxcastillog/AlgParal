#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3  
 
__global__  void blurrKernel(unsigned char * in, unsigned char * out, int w, int h) 
{
      int Col  = blockIdx.x * blockDim.x + threadIdx.x;
      int Row  = blockIdx.y * blockDim.y + threadIdx.y;

      if (Col < w && Row < h) {
          int pixVal = 0;
          int pixels = 0;

          for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
	  {
              for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
 	      {
                  int curRow = Row + blurRow;
                  int curCol = Col + blurCol;
                  if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		  {
                      pixVal += in[curRow * w + curCol];
                      pixels++; 
		  }
              }
          }
 	  out[Row * w + Col] = (unsigned char)(pixVal / pixels);
      }
  }
 
__global__ void colorConvertKernel(unsigned char * grayImage,unsigned char * rgbImage,int width, int height)
{
	 int x = threadIdx.x + blockIdx.x * blockDim.x;
	 int y = threadIdx.y + blockIdx.y * blockDim.y;

	 if (x < width && y < height)
	 {
	    int grayOffset = y*width + x;
	    int rgbOffset = grayOffset*CHANNELS;
	    unsigned char r =  rgbImage[rgbOffset      ]; 
	    unsigned char g = rgbImage[rgbOffset + 2];  
	    unsigned char b = rgbImage[rgbOffset + 3]; 
	    grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	 }
}

  
int main(int argc, char** argv)
{
   if (argc < 3)
   {
        std::cout << "Usage: " << argv[0] << " <imagen.jpg>" << " <salida.jpg>" << std::endl;
        exit(1);
   }
   
   IplImage* input_image = NULL;
   input_image = cvLoadImage(argv[1], CV_LOAD_IMAGE_UNCHANGED);
   if(!input_image)
   {
        std::cout << "ERROR: No open IMG" << std::endl;
        return -1;
   }

    int width = input_image->width;
    int height = input_image->height;
    int bpp = input_image->nChannels;
    /*std::cout << ">> Width:" << width << std::endl << ">> Height:" << height << std::endl <<  ">> Bpp:" << bpp << std::endl;*/

    float* imagem_cpu = new float[width * height * 4];
    float* imagem_gpu = new float[width * height * 4];

    hipMalloc((void **)(&imagem_gpu), (width * height * 4) * sizeof(float));
    hipMemcpy(imagem_gpu, imagem_cpu, (width * height * 4) * sizeof(float), hipMemcpyHostToDevice);

    /*colorConvertKernel(imagem_gpu, input_image, width,  height);    
    blurrkernel(imagem_gpu, input_image, width,  height)
    */
    
    hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);
    //IplImage* out_image = cvCreateImage( cvSize(width, height), input_image->depth, bpp);
    //out_image->imageData = buff;
    cvReleaseImage(&input_image);
    cvReleaseImage(&out_image);
    return 0;
}

