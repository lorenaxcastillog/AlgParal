#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3  
 
 
__global__ void colorConvertKernel(unsigned char * greyImage,unsigned char * rgbImage,int width, int height)
{
	 int Col = threadIdx.x + blockIdx.x * blockDim.x;
	 int Row = threadIdx.y + blockIdx.y * blockDim.y;

	 if (Col < width && Row < height)
	 {
	    int greyOffset = Row*width + Col;
	    int rgbOffset = greyOffset*CHANNELS;
	    unsigned char r =  rgbImage[rgbOffset   ]; 
	    unsigned char g = rgbImage[rgbOffset + 2];  
	    unsigned char b = rgbImage[rgbOffset + 3]; 
	    greyImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	 }
}

//1px
__global__  void blurrKernel(unsigned char * in, unsigned char * out, int w, int h) 
{
      int Col  = blockIdx.x * blockDim.x + threadIdx.x;
      int Row  = blockIdx.y * blockDim.y + threadIdx.y;

      if (Col < w && Row < h) 
      {
          int pixVal = 0;
          int pixels = 0;

//promedio de blur_size x blur_size box
          for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
	  {
              for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
 	      {
                  int curRow = Row + blurRow;
                  int curCol = Col + blurCol;
//verifica pixel
                  if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		  {
                      pixVal += in[curRow * w + curCol];
                      pixels++; 
		  }
              }
          }
 	  out[Row * w + Col] = (unsigned char)(pixVal / pixels);
      }
  }

  
int main(int argc, char** argv)
{
   IplImage* input_image = NULL; //datos imagen
   input_image = cvLoadImage(argv[1], CV_LOAD_IMAGE_UNCHANGED);
  
    int width = input_image->width;
    int height = input_image->height; 

    float* imagen_cpu = new float[width * height * 4];
    float* imagen_gpu = new float[width * height * 4];

    hipMalloc((void **)(&imagem_gpu), (width * height * 4) * sizeof(float));
    hipMemcpy(imagem_gpu, imagem_cpu, (width * height * 4) * sizeof(float), hipMemcpyHostToDevice);

//16Thrd
    dim3 dimGrid(ceil(width/16.0), ceil(height/16.0), 1);
    dim3 dimBlock(16, 16, 1);

    colorConvertKernel<<<dimGrid,dimBlock>>>(imagen_gpu, input_image, width,  height);
    
    blurrkernel<<<dimGrid,dimBlock>>>(imagem_gpu, input_image, width,  height)
    
    
    hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);
    cvReleaseImage(&input_image);
    cvReleaseImage(&out_image);
    return 0;
}

