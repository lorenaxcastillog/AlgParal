#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

///usr/local/bin/nvcc suma-matrices.cu -o suma.out

__global__ MatrizAddKernel_B(float *A, float* B, float* C, int n)
{
  int i=threadIdx.x+blockDim.x*blockIdx.x;
  if(i < n*n)
  {  
    C[i]=A[i]+B[i];
  }
}


__global__ MatrizAddKernel_C(float *A, float* B, float* C, int n)
{
  int i= n*blockIdx.x;
  if(i < n*n)
  {  
    for(int j = 0 ; j < n ; j++)
    {
      C[i + j]=A[i + j]+B[i + j];
    }
  }
}

__global__ MatrizAddKernel_D(float *A, float* B, float* C, int n)
{
  int i= blockIdx.x;
  if(i < n*n)
  {  
    for(int j = 0 ; j < n ; j++)
    {
      C[i+n *j]=A[i+n *j]+B[i+n *j];
    }
  }
}


void MatrizAdd(float *A, float *B, float *C, int n)
{
  float *d_A, *d_B, *d_C; 
  size_t size = n*n * sizeof(float);

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  MatrizAddKernel_B<<< 5 , 2 >>>(d_A, d_B, d_C n);	
  //MatrizAddKernel_C<<< n , 1 >>>(d_A, d_B, d_C, n);
  //MatrizAddKernel_D<<< n , 1 >>>(d_A, d_B, d_C, n);
  
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main(void)
{
    float *A, *B , *C; 
    int n = 10 ; 
    A = new float[n*n];
    B = new float[n*n];
    C = new float[n*n];
    for (int i = 0; i < n*n; ++i)
    {
        A[i] = i;
        B[i] = i;
    }
    
    MatrizAdd(A, B, C ,n); 
 
    for(int i = 0; i < n*n ; ++i)
    {
       cout << C[i] << " ";

    }
  return 0;
}
