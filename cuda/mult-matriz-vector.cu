#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void MultMatrizVectKernel(float *A, float *B, float *C, int n)
{
  int i = n * blockIdx.x; 
  if(i < n*n)
  {
    for(int j = 0; j < n ; ++j)
    {
      C[blockIdx.x] += A[i + j] * B[j];
    }
  }
} 

void MultMatrizVector(float *A, float *B, float *C, int n)
{
  float *d_A, *d_B, *d_C; 
  size_t size_A = n*n * sizeof(float);
  size_t size_B = n * sizeof(float);
  size_t size_C = n * sizeof(float);

  hipMalloc((void **) &d_A, size_A);
  hipMalloc((void **) &d_B, size_B);
  hipMalloc((void **) &d_C, size_C);

  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

  MultMatrizVectKernel<<< n, 1 >>>(d_A, d_B, d_C, n);

  hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main(void)
{
    float *A, *B , *C; 
    int n = 10 ; 
    A = new float[n*n];
    B = new float[n];
    C = new float[n];
    for (int i = 0; i < n*n; ++i)
    {
        A[i] = i;
    }

    for (int i = 0; i < n; ++i)
    {
        B[i] = i;
    }
    
    MultMatrizVector(A, B, C ,n); 
 
    for(int i = 0; i < n ; ++i)
    {
       cout << C[i] << " ";

    }
  return 0;
}
